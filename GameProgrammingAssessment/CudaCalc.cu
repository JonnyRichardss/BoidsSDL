#include "hip/hip_runtime.h"
#include "CudaCalc.cuh"
#include "hip/hip_runtime.h"
#include ""

#include "Global_Flags.h"
#include "Boid.h"
int NUMBLOCKS = 1024;
int NUMTHREADS = 1024;
//I previously tried converting these to half2 so that I could do vector operations on them but I can't figure out how to convert to half2
//weird results pop out of the builtin conversion functions
struct CudaBoidStruct {
	int id;
	float2 position;
	float2 velocity;
	float2 sepOutput = {0,0};
	float2 aligOutput = { 0,0 };
	float2 cohesOutput = { 0,0 };
	int numNeighbours = 0;
};
//(as from my previous AMP code) most of this adapted from
//https://github.com/SebLague/Boids/blob/master/Assets/Scripts/BoidCompute.compute
__global__ void GPUDoCalc(CudaBoidStruct* boids, int size, float sqrVisDist, float sqrAvoidDist) {
	int itest = blockIdx.x;
	int jtest = threadIdx.x;
	//int jtest = threadIdx.x;
	//for (int i = 0; i < size; i++) {
	//for (int jtest = 0; jtest < size; jtest++) {
			//if (i == j) continue;
		if (boids[itest].id == boids[jtest].id) return;
		float2 offset = make_float2(boids[itest].position.x - boids[jtest].position.x, boids[itest].position.y - boids[jtest].position.y);
		float sqrDist = offset.x * offset.x + offset.y * offset.y;
		if (sqrDist < sqrVisDist) {
			atomicAdd(&(boids[itest].numNeighbours),1);
			/*boids[itest].numNeighbours++;
			boids[itest].aligOutput.x += boids[jtest].velocity.x;
			boids[itest].aligOutput.y += boids[jtest].velocity.y;
			boids[itest].cohesOutput.x += boids[jtest].position.x;
			boids[itest].cohesOutput.y += boids[jtest].position.y;*/
			
			atomicAdd(&(boids[itest].aligOutput.x), boids[jtest].velocity.x);
			atomicAdd(&(boids[itest].aligOutput.y), boids[jtest].velocity.y);
			atomicAdd(&(boids[itest].cohesOutput.x), -boids[jtest].position.x);
			atomicAdd(&(boids[itest].cohesOutput.y), -boids[jtest].position.y);
			if (sqrDist < sqrAvoidDist) {
						/*boids[itest].sepOutput.x -= offset.x;
						boids[itest].sepOutput.y -= offset.y;*/
				atomicAdd(&(boids[itest].sepOutput.x), -offset.x);
				atomicAdd(&(boids[itest].sepOutput.y), -offset.y);
			}
			printf("Boid %i: added boid %i pos:%f,%f vel:%f,%f\n", boids[itest].id, boids[jtest].id, boids[jtest].position.x, boids[jtest].position.y, boids[jtest].velocity.x, boids[jtest].velocity.y);
		//}
	}
}
namespace JRCudaCalc {

	void MakeStructs(CudaBoidStruct* output, std::vector<Boid*>& input) {
		int size = input.size();
		for (int i = 0; i < size; i++) {
			Vector2 position = input[i]->GetPos();
			Vector2 velocity = input[i]->GetVelo();
			output[i].id = i;
			output[i].position = make_float2(position.x, position.y);
			output[i].velocity = make_float2(velocity.x, velocity.y);
			/*output[i].sepOutput = make_float2(0, 0);
			output[i].aligOutput = make_float2(0, 0);
			output[i].cohesOutput = make_float2(0, 0);
			output[i].numNeighbours = 0;*/
		}
	}
	void UnMakeStructs(std::vector<Boid*>& output, CudaBoidStruct* input) {
		int size = output.size();
		for (int i = 0; i < size; i++) {
			output[i]->steerTarget = Vector2::zero();
			output[i]->numNeighbours = input[i].numNeighbours;
			output[i]->DoSeparation(Vector2(input[i].sepOutput.x, input[i].sepOutput.y));
			output[i]->DoAlignment(Vector2(input[i].aligOutput.x, input[i].aligOutput.y));
			output[i]->DoCohesion(Vector2(input[i].cohesOutput.x, input[i].cohesOutput.y));
			output[i]->hasNeighbours = true;
		}
	}
	void DoCalc(std::vector<Boid*>& AllBoids)
	{
		GameLogging::GetInstance()->DebugLog("OI");
		int size = AllBoids.size();
		CudaBoidStruct* boids = new CudaBoidStruct[size];
		MakeStructs(boids, AllBoids);
		//allocate and copy
		CudaBoidStruct* gpuBoids;
		size_t arraySize = sizeof(CudaBoidStruct) * size;
		hipMalloc((void**)&gpuBoids, arraySize);
		hipMemcpy(gpuBoids, boids, arraySize, hipMemcpyHostToDevice);
		//do calc
		float sqrVisDist = BOID_VISION_DISTANCE * BOID_VISION_DISTANCE;
		float sqrAvoidDist = BOID_AVOID_DISTANCE * BOID_AVOID_DISTANCE;
		GPUDoCalc <<<NUMBLOCKS, NUMTHREADS>>> (gpuBoids, size,sqrVisDist , sqrAvoidDist);
		//copy back
		hipMemcpy(boids, gpuBoids, arraySize, hipMemcpyDeviceToHost);
		UnMakeStructs(AllBoids, boids);
		//free all
		hipFree(gpuBoids);
		delete[size] boids;
	}
}

